#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>
#include <fcntl.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char* file, int line, bool abort=true){
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__global__ void kernel(int* matrix, int* res_matrix, int size){
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int column = blockDim.y * blockIdx.y + threadIdx.y;
    if (row < size && column < size){
        int sum = 0;
        for (int rank = 0; rank < size; rank++)
            sum += matrix[row * size + rank] * matrix[rank * size + column];
        res_matrix[row * size + column] = sum;
    }
}
extern "C"  void multiplyMatrix(int* matrix, int* res_matrix, int size){
    int* cuda_matrix;
    gpuErrchk( hipMalloc((void**)&cuda_matrix, size * size * sizeof(int)) );
    int* cuda_res_matrix;
    gpuErrchk( hipMalloc((void**)&cuda_res_matrix, size * size * sizeof(int)) );
    gpuErrchk( hipMemcpy(cuda_matrix, matrix, size * size * sizeof(int), hipMemcpyHostToDevice) );
    dim3 threadsPerBlock(size, size);
    dim3 numBlocks(size / threadsPerBlock.x, size / threadsPerBlock.y);
    kernel<<<numBlocks, threadsPerBlock>>>(cuda_matrix, cuda_res_matrix, size);
    gpuErrchk( hipMemcpy(res_matrix, cuda_res_matrix, size * size * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk( hipFree(cuda_matrix) );
    gpuErrchk( hipFree(cuda_res_matrix) );
}
